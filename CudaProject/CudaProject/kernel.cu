#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include<iostream>
#include <hip/hip_runtime.h>

#include "kernel.cuh"

#include "ParticleSystem.h"

#define N 256

// Kernel Definition
__global__ void iter(Particle* p, Geometry g, int n)
{
	double NewXCoord, NewYCoord, NewVX, NewVY;
	double SourceCoordX_1 = 100;
	double SourceCoordY_1 = 200;
	double BasketLevel = 400;
	double Life = 1000;
	double dt = 1;
	int BasketCounter = 0;

	int i = threadIdx.x;

	if (i < n) {
		

		NewXCoord = p[i].GetCoords()[0] + p[i].GetVelocity()[0] * dt;
		NewYCoord = p[i].GetCoords()[1] + p[i].GetVelocity()[1] * dt;
		NewVX = p[i].GetVelocity()[0];
		NewVY = p[i].GetVelocity()[1];

		p[i].UpdateParticle(NewXCoord, NewYCoord, g, dt);

		if (p[i].GetLifetime() <= 0)
			p[i].UpdateLifeStatus(SourceCoordX_1, SourceCoordY_1, 0.01 * (rand() % 101), 0.01 * (rand() % 101), 0, 0.01 * (rand() % 101), 0, Life);

		if (p[i].GetCoords()[1] > BasketLevel) {
			BasketCounter += 1;
			p[i].UpdateLifeStatus(SourceCoordX_1, SourceCoordY_1, 0.01 * (rand() % 101), 0.01 * (rand() % 101), 0, 0.01 * (rand() % 101), 0, Life);
		}


	}
}

void Calc(Particle* h_a) {

	int i, j;
	double NewXCoord, NewYCoord, NewVX, NewVY;

	Geometry geom;

	// Allocate host memory

	// Initialize host array

	// Allocate arrays in Device memory
	Particle* d_a;
	Geometry g;
	
	hipMalloc((void**)&d_a, MAX_PARTICLES * sizeof(Particle));
//	hipMalloc((void)geom, sizeof(Geometry));

	// Copy memory from Host to Device
	hipMemcpy(d_a, h_a, MAX_PARTICLES * sizeof(Particle), hipMemcpyHostToDevice);
//	hipMemcpy(g, geom, sizeof(Geometry), hipMemcpyHostToDevice);

	// Block and Grid dimentions
	dim3 grid_size(1); dim3 block_size(N);

	// Launch Kernel
	iter << <grid_size, block_size >> > (d_a, g, N);

	// Some kind of synchronization
	hipDeviceSynchronize();

	hipMemcpy(h_a, d_a, N * sizeof(int), hipMemcpyDeviceToHost);


	for (int i = 0; i < 10; ++i) {
		//		printf("c[%d] = %d\n", i, h_a[i]);
		//		printf("c[%d] = %d\n", i, h_b[i]);
		std::cout << "h_a: c[" << i << "] = " << h_a[i] << "\n";

	}



//	free(h_a);
	hipFree(d_a);

	//	return 0;
}

/*
// Kernel Definition
__global__ void iter(int* a, int* b, int n)
{
	int i = threadIdx.x;
	if (i < n) {
		a[i] = a[i] * 2;
		b[i] = a[i] + 1;
	}
}

//void CalcFunction();

// int main() {
void Calc() {

	int* h_a;
	int* h_b;
	// Allocate host memory
	h_a = (int*)malloc(sizeof(int) * N);
	h_b = (int*)malloc(sizeof(int) * N);

	// Initialize host array
	for (int i = 0; i < N; i++) {
		h_a[i] = i;
		h_b[i] = i;
	}

	// Allocate arrays in Device memory
	int* d_a;
	int* d_b;
	hipMalloc((void**)& d_a, N * sizeof(int));
	hipMalloc((void**)& d_b, N * sizeof(int));

	// Copy memory from Host to Device
	hipMemcpy(d_a, h_a, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, N * sizeof(int), hipMemcpyHostToDevice);

	// Block and Grid dimentions
	dim3 grid_size(1); dim3 block_size(N);

	// Launch Kernel
	iter << <grid_size, block_size >> > (d_a, d_b, N);

	// Some kind of synchronization
	hipDeviceSynchronize();

	hipMemcpy(h_a, d_a, N * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(h_b, d_b, N * sizeof(int), hipMemcpyDeviceToHost);

	for (int i = 0; i < 10; ++i) {
		//		printf("c[%d] = %d\n", i, h_a[i]);
		//		printf("c[%d] = %d\n", i, h_b[i]);
		std::cout << "h_a: c[" << i << "] = " << h_a[i] << "\n";
		std::cout << "h_b: c[" << i << "] = " << h_b[i] << "\n";
	}

	free(h_a);
	free(h_b);
	hipFree(d_a);
	hipFree(d_b);

	//	return 0;
}
*/