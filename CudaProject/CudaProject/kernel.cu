#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_cooperative_groups.h"
#include <hip/device_functions.h>
#include <cuda_device_runtime_api.h>

#include <stdio.h>

#include<iostream>
#include <hip/hip_runtime.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "kernel.cuh"


//#include "ParticleSystem.h"
#include "Particle.h"
#include "Functions.h"
#include <algorithm>


__inline__ __device__ int warpReduceSum(int val)
{
	for (int offset = warpSize / 2; offset > 0; offset /= 2)
		val += __shfl_down_sync(warpSize - 1, val, offset);

	return val;
}

__inline__ __device__ int blockReduceSum(int val)
{
	static __shared__ int shared[32];
	int lane = threadIdx.x % warpSize;
	int wid = threadIdx.x / warpSize;

	val = warpReduceSum(val);

	// write reduced value to shared memory
	if (lane == 0)
		shared[wid] = val;

	__syncthreads();

	// ensure we only grab a value from shared memory 
	// if that warp existed
	val = (threadIdx.x < blockDim.x / warpSize) ? shared[lane] : int(0);

	if (wid == 0)
		val = warpReduceSum(val);

	return val;
}

__global__ void deviceReduceKernel(int* in, int* out, int n)
{
	int sum = 0;

	//reduce multiple elements per thread
	for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x)
		sum += in[i];

	sum = blockReduceSum(sum);

	if (threadIdx.x == 0)
		out[blockIdx.x] = sum;
}

void deviceReduce(int* in, int* out, int n)
{
	deviceReduceKernel << <BLOCKS_NUMBER, THREADS_NUMBER >> > (in, out, n);
	deviceReduceKernel << <1, THREADS_NUMBER >> > (out, out, BLOCKS_NUMBER);
}

// Kernel Definition
__global__ void iter(Particle* p1, Particle* p2, Geometry g, int n)
{
	double NewXCoord, NewYCoord;
	double SourceCoordX_1 = 300;
	double SourceCoordY_1 = 100;
	double SourceCoordX_2 = 500;
	double SourceCoordY_2 = 100;
	double BasketLevel = 400;
	double BasketBegin = 350;
	double BasketWidth = 100;
	double Life = 10000;
	double dt = 0.5;

	ParticleParams params1;
	ParticleParams params2;
	int b;

	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < n) {
		
		params1 = p1[i].GetParams();
		NewXCoord = params1.x + params1.Vx * dt;
		NewYCoord = params1.y + params1.Vy * dt;

		p1[i].UpdateParticle(NewXCoord, NewYCoord, g, dt);
		if (params1.Life <= 0) {	
			p1[i].UpdateLifeStatus(SourceCoordX_1, SourceCoordY_1, 0.1, 0.1, 0, 1, 0, Life);
		}

		if (params1.y > BasketLevel) {
			p1[i].UpdateLifeStatus(SourceCoordX_1, SourceCoordY_1, 0.1, 0.1, 0, 1, 0, Life);
		}

		if (params1.y >= BasketLevel && params1.x >= BasketBegin && params1.x <= BasketBegin + BasketWidth) {
			b = params1.InBasket + 1;
			p1[i].SetInBasket(b);
		}

		params2 = p2[i].GetParams();
		NewXCoord = params2.x + params2.Vx * dt;
		NewYCoord = params2.y + params2.Vy * dt;

		p2[i].UpdateParticle(NewXCoord, NewYCoord, g, dt);

		if (params2.Life <= 0) {
			p2[i].UpdateLifeStatus(SourceCoordX_2, SourceCoordY_2, 0.1, 0.1, 0, 0, 1, Life);
		}

		if (params2.y > BasketLevel) {
			p2[i].UpdateLifeStatus(SourceCoordX_2, SourceCoordY_2, 0.1, 0.1, 0, 0, 1, Life);
		}

		if (params2.y > BasketLevel && params2.x >= BasketBegin && params2.x <= BasketBegin + BasketWidth) {
			b = params2.InBasket + 1;
			p2[i].SetInBasket(b);
		}

	}

	__syncthreads();
}

__global__ void iter2(Particle* p1, Particle* p2, int n) {

	double tempVx, tempVy;
	double ParticlesDist;
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j = blockDim.y * blockIdx.y + threadIdx.y;

	// Checking particle-particle interaction

	if (i >= n || j >= n)
		return;

	ParticlesDist = RDistance(p1[i].GetParams().x, p2[j].GetParams().x, p1[i].GetParams().y, p2[j].GetParams().y);

	if (ParticlesDist <= 14.0) {

		tempVx = p1[i].GetParams().Vx;
		tempVy = p1[i].GetParams().Vy;

		p1[i].SetVelocity(p2[j].GetParams().Vx, p2[j].GetParams().Vy);
		p2[j].SetVelocity(tempVx, tempVy);

	}

		__syncthreads();
	
}

void Calc(Particle* h_a, Particle* h_b, Geometry g, int n) {


	// Allocate host memory
	// Initialize host array

	// Allocate arrays in Device memory
	Particle* d_a;
	Particle* d_b;

	size_t size= BLOCKS_NUMBER*THREADS_NUMBER*sizeof(Particle);

	hipMalloc(&d_a, size);
	hipMalloc(&d_b, size);

	// Copy memory from Host to Device
	hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

	// Block and Grid dimentions
	
	iter<<<BLOCKS_NUMBER,THREADS_NUMBER>>>(d_a, d_b, g, n);
	// Launch Kernel

	// Some kind of synchronization
	hipDeviceSynchronize();

	hipMemcpy(h_a, d_a, size, hipMemcpyDeviceToHost);
	hipMemcpy(h_b, d_b, size, hipMemcpyDeviceToHost);

	hipFree(d_a);
	hipFree(d_b);
}

void Calc2(Particle* h_a, Particle* h_b, int n) {

	// Allocate host memory
// Initialize host array

// Allocate arrays in Device memory
	Particle* d_a;
	Particle* d_b;

	size_t size = BLOCKS_NUMBER * THREADS_NUMBER * sizeof(Particle);

	hipMalloc(&d_a, size);
	hipMalloc(&d_b, size);

	// Copy memory from Host to Device
	hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

	// Block and Grid dimentions

	dim3 block(BLOCKS_NUMBER, BLOCKS_NUMBER);
	dim3 grid(THREADS_NUMBER, THREADS_NUMBER);
	iter2 << <grid, block >> > (d_a, d_b, n);
	// Launch Kernel

	// Some kind of synchronization
	hipDeviceSynchronize();

	hipMemcpy(h_a, d_a, size, hipMemcpyDeviceToHost);
	hipMemcpy(h_b, d_b, size, hipMemcpyDeviceToHost);

	hipFree(d_a);
	hipFree(d_b);
}

int Calc3(Particle* h_a, Particle* h_b, int n) {

	// Allocate host memory
// Initialize host array

// Allocate arrays in Device memory
	int counter = 0;
	int* a;
	int* b;

	// allocate memory
	hipMallocManaged(&a, 2 * n * sizeof(int));
	hipMallocManaged(&b, 2 * n * sizeof(int) / THREADS_NUMBER);	// we need space for every block, ie n/512 elements

		// fill it with data
	for (int i = 0; i < n; i++) {
		a[i] = h_a[i].GetInBasket();
		a[i + n] = h_b[i].GetInBasket();
	}

	deviceReduce(a, b, 2*n);

	hipDeviceSynchronize();

	counter = b[0];

	hipFree(a);
	hipFree(b);

	return counter;
}
