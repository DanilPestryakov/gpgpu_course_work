#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include<iostream>
#include <hip/hip_runtime.h>

#include "kernel.cuh"

#include "ParticleSystem.h"


// Kernel Definition
__global__ void iter(Particle* p, Geometry g, int n)
{
	double NewXCoord, NewYCoord, NewVX, NewVY;
	double SourceCoordX_1 = 100;
	double SourceCoordY_1 = 200;
	double BasketLevel = 400;
	double Life = 1000;
	double dt = 1;
	int BasketCounter = 0;

	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < n) {
		

		NewXCoord = p[i].GetCoords()[0] + p[i].GetVelocity()[0] * dt;
		NewYCoord = p[i].GetCoords()[1] + p[i].GetVelocity()[1] * dt;
		NewVX = p[i].GetVelocity()[0];
		NewVY = p[i].GetVelocity()[1];

		p[i].UpdateParticle(NewXCoord, NewYCoord, g, dt);

		if (p[i].GetLifetime() <= 0)
			p[i].UpdateLifeStatus(SourceCoordX_1, SourceCoordY_1, 0.01 * (rand() % 101), 0.01 * (rand() % 101), 0, 0.01 * (rand() % 101), 0, Life);

		if (p[i].GetCoords()[1] > BasketLevel) {
			BasketCounter += 1;
			p[i].UpdateLifeStatus(SourceCoordX_1, SourceCoordY_1, 0.01 * (rand() % 101), 0.01 * (rand() % 101), 0, 0.01 * (rand() % 101), 0, Life);
		}


	}
}

void Calc(Particle* h_a, Geometry g, int n) {

	int i, j;
	double NewXCoord, NewYCoord, NewVX, NewVY;

	Geometry geom;

	// Allocate host memory

	// Initialize host array

	// Allocate arrays in Device memory
	Particle* d_a;
	size_t size= BLOCKS_NUMBER*THREADS_NUMBER*sizeof(Particle);

	hipMalloc(&d_a, size);

	// Copy memory from Host to Device
	hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
//	hipMemcpy(g, geom, sizeof(Geometry), hipMemcpyHostToDevice);

	// Block and Grid dimentions
	
	iter<<<BLOCKS_NUMBER,THREADS_NUMBER>>>(dev_classarray);
	// Launch Kernel
	iter << <grid_size, block_size >> > (d_a, g, n);

	// Some kind of synchronization
	hipDeviceSynchronize();

	hipMemcpy(h_a, d_a, size, hipMemcpyDeviceToHost);


//	free(h_a);
	hipFree(d_a);

	//	return 0;
}