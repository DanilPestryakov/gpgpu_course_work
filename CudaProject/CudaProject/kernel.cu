#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include<iostream>
#include <hip/hip_runtime.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "kernel.cuh"

#include "ParticleSystem.h"
#include "Particle.h"

// Kernel Definition
__global__ void iter(Particle* p, Geometry g, int n)
{
	double NewXCoord, NewYCoord, NewVX, NewVY;
	double SourceCoordX_1 = 100;
	double SourceCoordY_1 = 200;
	double BasketLevel = 400;
	double Life = 1000;
	double dt = 1;
	int BasketCounter = 0;

	Coords coords;

	hiprandState_t state;
	double numb;
	double RANGE = 1;

	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < n) {
		
		coords = p[i].GetCoords();
		NewXCoord = coords.x + coords.Vx * dt;
		NewYCoord = coords.y + coords.Vy * dt;
		NewVX = coords.Vx;
		NewVY = coords.Vy;

		p[i].UpdateParticle(NewXCoord, NewYCoord, g, dt);

		if (coords.y <= 0) {
			numb = (double)((hiprand_uniform(&state)*(RANGE+1)));
			p[i].UpdateLifeStatus(SourceCoordX_1, SourceCoordY_1, numb, numb, 0, numb, 0, Life);
		}

		if (coords.y > BasketLevel) {
			BasketCounter += 1;
			numb = (double)((hiprand_uniform(&state) * (RANGE + 1)));
			p[i].UpdateLifeStatus(SourceCoordX_1, SourceCoordY_1, numb, numb, 0, numb, 0, Life);
		}


	}
}

void Particle::Calc(Particle* h_a, Geometry g, int n) {

	int i, j;
	double NewXCoord, NewYCoord, NewVX, NewVY;

//	Geometry geom;

	// Allocate host memory

	// Initialize host array

	// Allocate arrays in Device memory
	Particle* d_a;

	size_t size= BLOCKS_NUMBER*THREADS_NUMBER*sizeof(Particle);

	hipMalloc(&d_a, size);

	// Copy memory from Host to Device
	hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
//	hipMemcpy(g, geom, sizeof(Geometry), hipMemcpyHostToDevice);

	// Block and Grid dimentions
	
	iter<<<BLOCKS_NUMBER,THREADS_NUMBER>>>(d_a, g, n);
	// Launch Kernel
//	iter << <grid_size, block_size >> > (d_a, g, n);

	// Some kind of synchronization
	hipDeviceSynchronize();

	hipMemcpy(h_a, d_a, size, hipMemcpyDeviceToHost);


//	free(h_a);
	hipFree(d_a);

}