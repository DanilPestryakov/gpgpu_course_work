#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_cooperative_groups.h"
#include <hip/device_functions.h>
#include <cuda_device_runtime_api.h>

#include <stdio.h>

#include<iostream>
#include <hip/hip_runtime.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "kernel.cuh"

//#include "ParticleSystem.h"
#include "Particle.h"
#include "Functions.h"


// Kernel Definition
__global__ void iter(Particle* p1, Particle* p2, Geometry g, int n)
{
	double NewXCoord, NewYCoord, NewVX, NewVY;
	double SourceCoordX_1 = 300;
	double SourceCoordY_1 = 100;
	double SourceCoordX_2 = 500;
	double SourceCoordY_2 = 100;
	double BasketLevel = 400;
	double Life = 10000;
	double dt = 0.3;
	int BasketCounter = 0;

	ParticleParams params1;
	ParticleParams params2;

	hiprandState_t state;
	hiprand_init(0, 0, 0, &state);

	double numb;
	int RANGE = 1;
	int MAX = 101;

//	numb = hiprand_uniform(&state);
//	printf("%d\n", numb);

	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < n) {
		
		params1 = p1[i].GetParams();
		NewXCoord = params1.x + params1.Vx * dt;
		NewYCoord = params1.y + params1.Vy * dt;
		NewVX = params1.Vx;
		NewVY = params1.Vy;

		p1[i].UpdateParticle(NewXCoord, NewYCoord, g, dt);
		if (params1.Life <= 0) {
			numb = (double)((hiprand_uniform(&state)*(RANGE+1)));		
			p1[i].UpdateLifeStatus(SourceCoordX_1, SourceCoordY_1, 0.1, 0.1, 0, 1, 0, Life);
		}

		if (params1.y > BasketLevel) {
			BasketCounter += 1;
			numb = (double)((hiprand_uniform(&state) * (RANGE + 1)));
			p1[i].UpdateLifeStatus(SourceCoordX_1, SourceCoordY_1, 0.1, 0.1, 0, 1, 0, Life);
		}

		params2 = p2[i].GetParams();
		NewXCoord = params2.x + params2.Vx * dt;
		NewYCoord = params2.y + params2.Vy * dt;
		NewVX = params2.Vx;
		NewVY = params2.Vy;

		p2[i].UpdateParticle(NewXCoord, NewYCoord, g, dt);

		if (params2.Life <= 0) {
			numb = (double)((hiprand_uniform(&state) * (RANGE + 1)));
			p2[i].UpdateLifeStatus(SourceCoordX_2, SourceCoordY_2, 0.1, 0.1, 0, 0, 1, Life);
		}

		if (params2.y > BasketLevel) {
			BasketCounter += 1;
			numb = (double)((hiprand_uniform(&state) * (RANGE + 1)));
			p2[i].UpdateLifeStatus(SourceCoordX_2, SourceCoordY_2, 0.1, 0.1, 0, 0, 1, Life);
		}

	}

	__syncthreads();
}

__global__ void iter2(Particle* p1, Particle* p2, int n) {

	double tempVx, tempVy;
	double ParticlesDist;
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j = blockDim.y * blockIdx.y + threadIdx.y;

	// Checking particle-particle interaction

	if (i >= n || j >= n)
		return;

	ParticlesDist = RDistance(p1[i].GetParams().x, p2[j].GetParams().x, p1[i].GetParams().y, p2[j].GetParams().y);

	if (ParticlesDist <= 14.0) {

		tempVx = p1[i].GetParams().Vx;
		tempVy = p1[i].GetParams().Vy;

		p1[i].SetVelocity(p2[j].GetParams().Vx, p2[j].GetParams().Vy);
		p2[j].SetVelocity(tempVx, tempVy);

	}

		__syncthreads();
	
}

void Calc(Particle* h_a, Particle* h_b, Geometry g, int n) {

	// Allocate host memory
	// Initialize host array

	// Allocate arrays in Device memory
	Particle* d_a;
	Particle* d_b;

	size_t size= BLOCKS_NUMBER*THREADS_NUMBER*sizeof(Particle);

	hipMalloc(&d_a, size);
	hipMalloc(&d_b, size);

	// Copy memory from Host to Device
	hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

	// Block and Grid dimentions
	
	iter<<<BLOCKS_NUMBER,THREADS_NUMBER>>>(d_a, d_b, g, n);
	// Launch Kernel

	// Some kind of synchronization
	hipDeviceSynchronize();

	hipMemcpy(h_a, d_a, size, hipMemcpyDeviceToHost);
	hipMemcpy(h_b, d_b, size, hipMemcpyDeviceToHost);

	hipFree(d_a);
	hipFree(d_b);

}

void Calc2(Particle* h_a, Particle* h_b, int n) {
	// Allocate host memory
// Initialize host array

// Allocate arrays in Device memory
	Particle* d_a;
	Particle* d_b;

	size_t size = BLOCKS_NUMBER * THREADS_NUMBER * sizeof(Particle);

	hipMalloc(&d_a, size);
	hipMalloc(&d_b, size);

	// Copy memory from Host to Device
	hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

	// Block and Grid dimentions

	dim3 block(BLOCKS_NUMBER, BLOCKS_NUMBER);
	dim3 grid(THREADS_NUMBER, THREADS_NUMBER);
	iter2 << <grid, block >> > (d_a, d_b, n);
	// Launch Kernel

	// Some kind of synchronization
	hipDeviceSynchronize();

	hipMemcpy(h_a, d_a, size, hipMemcpyDeviceToHost);
	hipMemcpy(h_b, d_b, size, hipMemcpyDeviceToHost);

	hipFree(d_a);
	hipFree(d_b);
}