#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include<iostream>
#include <hip/hip_runtime.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "kernel.cuh"

//#include "ParticleSystem.h"
#include "Particle.h"

// Kernel Definition
__global__ void iter(Particle* p, Geometry g, int n)
{
	double NewXCoord, NewYCoord, NewVX, NewVY;
	double SourceCoordX_1 = 100;
	double SourceCoordY_1 = 200;
	double BasketLevel = 400;
	double Life = 1000;
	double dt = 1;
	int BasketCounter = 0;

	ParticleParams params;

	hiprandState_t state;
	double numb;
	double RANGE = 1;

	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < n) {
		
		params = p[i].GetParams();
		NewXCoord = params.x + params.Vx * dt;
		NewYCoord = params.y + params.Vy * dt;
		NewVX = params.Vx;
		NewVY = params.Vy;

		p[i].UpdateParticle(NewXCoord, NewYCoord, g, dt);

		if (params.y <= 0) {
			numb = (double)((hiprand_uniform(&state)*(RANGE+1)));
			p[i].UpdateLifeStatus(SourceCoordX_1, SourceCoordY_1, numb, numb, 0, numb, 0, Life);
		}

		if (params.y > BasketLevel) {
			BasketCounter += 1;
			numb = (double)((hiprand_uniform(&state) * (RANGE + 1)));
			p[i].UpdateLifeStatus(SourceCoordX_1, SourceCoordY_1, numb, numb, 0, numb, 0, Life);
		}


	}
}

void Calc(Particle* h_a, Geometry g, int n) {

	int i, j;
	double NewXCoord, NewYCoord, NewVX, NewVY;

//	Geometry geom;

	// Allocate host memory

	// Initialize host array

	// Allocate arrays in Device memory
	Particle* d_a;

	size_t size= BLOCKS_NUMBER*THREADS_NUMBER*sizeof(Particle);

	hipMalloc(&d_a, size);

	// Copy memory from Host to Device
	hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
//	hipMemcpy(g, geom, sizeof(Geometry), hipMemcpyHostToDevice);

	// Block and Grid dimentions
	
	iter<<<BLOCKS_NUMBER,THREADS_NUMBER>>>(d_a, g, n);
	// Launch Kernel
//	iter << <grid_size, block_size >> > (d_a, g, n);

	// Some kind of synchronization
	hipDeviceSynchronize();

	hipMemcpy(h_a, d_a, size, hipMemcpyDeviceToHost);


//	free(h_a);
	hipFree(d_a);

}